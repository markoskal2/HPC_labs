#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define maxAccuracy  	0.00000005
#define minAccuracy	5.00

#define COMMAND_INPUT
#define TIME_CALC
#define ACC

// #define CPU_COMPARISON
#define TILED_CONVOLUTION
#define FILTER_RADIUS 16

/*******************CUDA Error Check*******************/
#define CUDAsafeCall(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define CUDAcheckError() __cudaCheckError(__FILE__, __LINE__)

inline void gpuAssert(hipError_t err, const char *file, int line, bool abort = true) {

    if(err != hipSuccess) {
	fprintf(stderr, "GPUAssert: %s %s %d\n", hipGetErrorString(err), file, line);
	if(abort)
	    exit(err);
    }
}

inline void __cudaCheckError(const char *file, const int line) {

    hipError_t error = hipGetLastError();
    if(hipSuccess != error) {
		fprintf(stderr, "CUDAcheckError failed at %s: %i: %s\n", file, line, hipGetErrorString(error));
		exit(-1);
    }

    return; //an ola pane kala
}
/*******************CUDA Error Check*******************/

#ifdef TILED_CONVOLUTION
	__constant__ float const_filter[FILTER_RADIUS*FILTER_RADIUS];

	////////////////////////////////////////////////////////////////////////////////
	// TILED GPU row convolution filter
	////////////////////////////////////////////////////////////////////////////////
	__global__ void tiledconvolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, int row_width, int imageW, int imageH, int filterR, int tile_width, int block, int ratio) {
		extern __shared__ float data_shared[]; //pinakas pou tha fortwthoun ta dedomena sti shared

		int i, j, srcX, srcY;
		int k, d, colX, rowY;

		float sum;

		for(i = 0; i < ratio; i++) {
			for(j = 0; j < ratio; j++) {
				rowY = (i * tile_width)/ratio + threadIdx.y;
				srcY = blockIdx.y * block + rowY + filterR;
				
				colX = (j * tile_width)/ratio + threadIdx.x;
				srcX = blockIdx.x * block + colX + filterR;

				data_shared[rowY * (tile_width + 1) + colX] = d_Src[srcY * row_width + srcX];
				__syncthreads();
			}
		}


		for(i = 0; i < ratio; i++) {
			for(j = 0; j < ratio; j++) {
				rowY = (i * tile_width)/ratio + threadIdx.y;
				srcY = blockIdx.y * block + rowY + filterR;
				
				colX = (j * tile_width)/ratio + threadIdx.x;
				srcX = blockIdx.x * block + colX + filterR;

				sum = 0;

				for(k = -filterR; k <= filterR; k++) {
					d = colX + k;
					//an vriskomaste entos twn oriwn mporoume
					//na xrisimopoihsoume th shared memory
					if(d >= 0 && d < tile_width) {
						sum += data_shared[rowY * (tile_width + 1) + d] * const_filter[filterR - k];
					}
					else {
						sum += d_Src[srcY * row_width + srcX + k] * const_filter[filterR - k];
					}
				}
				d_Dst[srcY * row_width + srcX] = sum;
				__syncthreads();
			}
		}
	}

////////////////////////////////////////////////////////////////////////////////
// TILED GPU column convolution filter
////////////////////////////////////////////////////////////////////////////////
	__global__ void tiledconvolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter, int row_width, int imageW, int imageH, int filterR, int tile_width, int block, int ratio) {
		extern __shared__ float data_shared[]; //pinakas pou tha fortwthoun ta dedomena sti shared

		int i, j, srcX, srcY;
		int k, d, colX, rowY;

		float sum;

		for(i = 0; i < ratio; i++) {
			for(j = 0; j < ratio; j++) {
				rowY = (i * tile_width)/ratio + threadIdx.y;
				srcY = blockIdx.y * block + rowY + filterR;
				
				colX = (j * tile_width)/ratio + threadIdx.x;
				srcX = blockIdx.x * block + colX + filterR;

				data_shared[rowY * (tile_width + 1) + colX] = d_Src[srcY * row_width + srcX];
				__syncthreads();
			}
		}


		for(i = 0; i < ratio; i++) {
			for(j = 0; j < ratio; j++) {
				rowY = (i * tile_width)/ratio + threadIdx.y;
				srcY = blockIdx.y * block + rowY + filterR;
				
				colX = (j * tile_width)/ratio + threadIdx.x;
				srcX = blockIdx.x * block + colX + filterR;

				sum = 0;

				for(k = -filterR; k <= filterR; k++) {
					d = rowY + k;
					//an vriskomaste entos twn oriwn mporoume
					//na xrisimopoihsoume th shared memory
					if(d >= 0 && d < tile_width) {
						sum += data_shared[d * (tile_width + 1) + colX] * const_filter[filterR - k];
					}
					else {
						sum += d_Src[(srcY + k) * row_width + srcX] * const_filter[filterR - k];
					}
				}
				d_Dst[(srcY-filterR) * imageW + srcX - filterR] = sum;
				__syncthreads();
			}
		}
	}

#else

////////////////////////////////////////////////////////////////////////////////
// GPU row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, int row_width,
		       int imageW, int imageH, int filterR) {

    int blockID = (gridDim.x * blockIdx.y) + (gridDim.x * gridDim.y * blockIdx.z) + blockIdx.x;

    int threadID = threadIdx.x + (blockID * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x);

    int x = (threadID % imageH) + filterR;
    int y = (threadID / imageH) + filterR;
    int k, d;

    float sum = 0;

    for (k = -filterR; k <= filterR; k++) {
        d = x + k;
	    sum += d_Src[y * row_width + d] * d_Filter[filterR - k];
    }
    d_Dst[y * row_width + x] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// GPU column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter, int row_width,
				     int imageW, int imageH, int filterR) {

    int blockID = (gridDim.x * blockIdx.y) + (gridDim.x * gridDim.y * blockIdx.z) + blockIdx.x;

    int threadID = threadIdx.x + (blockID * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x);

    int x = (threadID % imageH) + filterR;
    int y = (threadID / imageH) + filterR;
    int k, d;

    float sum = 0;

    for (k = -filterR; k <= filterR; k++) {
        d = y + k;
	    sum += d_Src[d * row_width + x] * d_Filter[filterR - k];

    }
    d_Dst[(y-filterR) * imageH + x - filterR] = sum;
}

#endif

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter,
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU;


    int imageW;
    int imageH;
    unsigned int i;

#ifdef COMMAND_INPUT
    #ifdef TILED_CONVOLUTION
    	if(argc != 4) {
    		printf("Error with command line args.\n");
    		exit(-1);
    	}
    	filter_radius = (unsigned int) atoi(argv[1]);
    	imageW = atoi(argv[2]);
    	int ratio = atoi(argv[3]); //tile to block ratio

    	if(ratio > imageW) {
    		printf("Error, imageW cannot be smaller than ratio.\n");
    		exit(-1);
    	}
    #else
	    if(argc != 3) {
			printf("Error with command line args.\n");
			exit(1);
	    }
	    filter_radius = (unsigned int) atoi(argv[1]);
	    imageW = atoi(argv[2]);
	#endif    

#else

    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);

#endif

    imageH = imageW;
#ifndef ACC

    printf("Filter Radius: %i\nFilter Length: %i\n", filter_radius, FILTER_LENGTH);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);

#endif

    //printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));

    /*****************************CUDA*****************************/

    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
    *h_OutputGPU,
    *h_InputGPU;

    unsigned int row_width = filter_radius * 2 + imageH;
    unsigned int size = row_width*row_width * sizeof(float);

    h_OutputGPU = (float *) malloc(imageW * imageH * sizeof(float));
    h_InputGPU = (float *) malloc(size);

    CUDAsafeCall(hipMalloc((void **) &d_Filter, FILTER_LENGTH * sizeof(float)));
    CUDAsafeCall(hipMalloc((void **) &d_Input, size));
    CUDAsafeCall(hipMalloc((void **) &d_Buffer, size));
    CUDAsafeCall(hipMalloc((void **) &d_OutputGPU, imageW * imageH * sizeof(float)));

    /**************************************************************/

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < (unsigned int) imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    //printf("CPU computation...\n");

#ifdef CPU_COMPARISON
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
#endif

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

    /***************PADDING***************/
    memset(h_InputGPU, 0.0, size);
    CUDAsafeCall(hipMemset(d_Buffer, 0.0, size));

    unsigned int x, y;
    for(i = 0; i < (unsigned int) imageW * imageH; i++) {
		x = filter_radius + i % imageH;
		y = filter_radius + i / imageH;
		h_InputGPU[y * row_width + x] = h_Input[i];
    }

    /*****************************CUDA ALLOC*****************************/

    CUDAsafeCall(hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice));
    CUDAsafeCall(hipMemcpy(d_Input, h_InputGPU, size, hipMemcpyHostToDevice));

    /**************************************************************/

#ifdef TILED_CONVOLUTION
    hipMemcpyToSymbol(HIP_SYMBOL(const_filter), h_Filter, FILTER_LENGTH * sizeof(float));

    int tile_width, block;
    int blockDimX, gridDimX, shared_mem;

    blockDimX = imageW / ratio;
    if(blockDimX >= 32)
    	blockDimX = 32;

    tile_width = blockDimX * ratio;
    while(tile_width >= 128) {
    	blockDimX /= 2;
    	tile_width = blockDimX * ratio;
    }

    gridDimX = imageW / (blockDimX * ratio);
    block = blockDimX * ratio;

    shared_mem = tile_width * (tile_width + 1) * sizeof(float);

#ifndef ACC
    printf("Tile: %d, %d\ndimBlock: %d, %d\ndimGrid: %d,%d\nShared memory in Bytes: %d\n", tile_width, tile_width, blockDimX, blockDimX, gridDimX, gridDimX, shared_mem);
#endif

    dim3 dimBlock(blockDimX, blockDimX);
    dim3 dimGrid(gridDimX, gridDimX);

#ifdef TIME_CALC
    struct timespec tv1, tv2;
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
#endif

    tiledconvolutionRowGPU<<<dimGrid, dimBlock, shared_mem>>>(d_Buffer, d_Input, d_Filter, (int) row_width, imageW, imageH, filter_radius, tile_width, block, ratio);
    CUDAsafeCall(hipPeekAtLastError());
    CUDAsafeCall(hipDeviceSynchronize());

    tiledconvolutionRowGPU<<<dimGrid, dimBlock, shared_mem>>>(d_Buffer, d_Input, d_Filter, (int) row_width, imageW, imageH, filter_radius, tile_width, block, ratio);
    CUDAsafeCall(hipPeekAtLastError());
    CUDAsafeCall(hipDeviceSynchronize());

#else

    int blockDimX, gridDimX;
    if(imageW <=32)
    	blockDimX = imageW;
    else
    	blockDimX = 32;

    if((imageW*imageH/1024) > 0)
    	gridDimX = imageW*imageH/1024;
    else
    	gridDimX = 1;

    gridDimX = sqrt(gridDimX);
    if(gridDimX > 65535)
    	gridDimX = 65535;

    dim3 dimBlock(blockDimX, blockDimX);
    dim3 dimGrid(gridDimX, gridDimX);

#ifdef TIME_CALC
	//start
    struct timespec tv1, tv2;
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
#endif

    convolutionRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, (int) row_width, imageW, imageH, filter_radius);
    CUDAsafeCall(hipPeekAtLastError());
    CUDAsafeCall(hipDeviceSynchronize());

    convolutionColumnGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, (int) row_width, imageW, imageH, filter_radius);
    CUDAsafeCall(hipPeekAtLastError());
    CUDAsafeCall(hipDeviceSynchronize());
#endif

#ifdef TIME_CALC
    //stamata to roloi
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
    #ifdef ACC
		printf ("%10g \n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));
	#else
		printf ("GPU time: %10g seconds\n",
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec));
	#endif
#endif

	CUDAsafeCall(hipMemcpy(h_OutputGPU, d_OutputGPU, imageW*imageH*sizeof(float), hipMemcpyDeviceToHost));


#ifdef CPU_COMPARISON
    int err;
    float acc;
    for(acc = maxAccuracy; acc <= minAccuracy; acc *= 10) {
		err = 0;
		for(i = 0; i < (unsigned int) imageW*imageH; i++) {
			if(acc < ABS(h_OutputCPU[i] - h_OutputGPU[i])) {
				err = 1;
				break;
			}
		}

		if(err == 0) {
#ifndef ACC
			printf("Max Accuracy: %f\n", acc);
#endif
			break;
		}
    }
    if(err) {
#ifndef ACC
	printf("Image is not accurate with filter: %i x %i\n", filter_radius, filter_radius);
#endif
    }
#endif

    CUDAsafeCall(hipFree(d_Filter));
    CUDAsafeCall(hipFree(d_Buffer));
    CUDAsafeCall(hipFree(d_Input));
    CUDAsafeCall(hipFree(d_OutputGPU));

    free(h_OutputGPU);
    free(h_InputGPU);
    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    CUDAsafeCall(hipDeviceReset());

    return 0;
}
