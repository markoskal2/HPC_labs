/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define maxAccuracy  	0.00005
#define minAccuracy	5.00

#define COMMAND_INPUT

/*******************CUDA Error Check*******************/
#define CUDAsafeCall(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define CUDAcheckError() __cudaCheckError(__FILE__, __LINE__)

inline void gpuAssert(hipError_t err, const char *file, int line, bool abort = true) {
    
    if(err != hipSuccess) {
	fprintf(stderr, "GPUAssert: %s %s %d\n", hipGetErrorString(err), file, line);
	if(abort)
	    exit(err);
    }
}

inline void __cudaCheckError(const char *file, const int line) {
    
    hipError_t error = hipGetLastError();
    if(hipSuccess != error) {
	fprintf(stderr, "CUDAcheckError failed at %s: %i: %s\n", file, line, hipGetErrorString(error));
	exit(-1);
    }
    
    return; //an ola pane kala
}
/*******************CUDA Error Check*******************/

////////////////////////////////////////////////////////////////////////////////
// GPU row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, 
		       int imageW, int imageH, int filterR) {
    
    int x = threadIdx.x;
    int y = threadIdx.y;
    int k, d;
    
    float sum = 0;
    
    for (k = -filterR; k <= filterR; k++) {
	d = x + k;
	
	if (d >= 0 && d < imageW) {
	    sum += d_Src[y * imageW + d] * d_Filter[filterR - k];
	}     
	
	d_Dst[y * imageH + x] = sum;
    }
    
}

////////////////////////////////////////////////////////////////////////////////
// GPU column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter,
				     int imageW, int imageH, int filterR) {
    
    int x = threadIdx.x;
    int y = threadIdx.y; 
    int k, d;
    
    float sum = 0;
    
    for (k = -filterR; k <= filterR; k++) {
	d = y + k;
	
	if (d >= 0 && d < imageH) {
	    sum += d_Src[d * imageW + x] * d_Filter[filterR - k];
	}
	
	d_Dst[y * imageH + x] = sum;
    }
    
				     }
				     


////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU;


    int imageW;
    int imageH;
    unsigned int i;
    
#ifdef COMMAND_INPUT
    
    if(argc != 3) {
	printf("Error with commanda line args.");
	exit(1);
    }
    filter_radius = (unsigned int) atoi(argv[1]);
    imageW = atoi(argv[2]);
    
#else
    
    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    
#endif
    
    imageH = imageW;
    printf("Filter Radius: %d\nFilter Length: %d\n", filter_radius, FILTER_LENGTH);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    //printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    
    /*****************************CUDA*****************************/
    
    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
    *h_OutputGPU;
    
    h_OutputGPU = (float *) malloc(imageW * imageH * sizeof(float));

    CUDAsafeCall(hipMalloc((void **) &d_Filter, FILTER_LENGTH * sizeof(float)));
    CUDAsafeCall(hipMalloc((void **) &d_Input, imageW * imageH * sizeof(float)));
    CUDAsafeCall(hipMalloc((void **) &d_Buffer, imageW * imageH * sizeof(float)));
    CUDAsafeCall(hipMalloc((void **) &d_OutputGPU, imageW * imageH * sizeof(float)));
    
    /**************************************************************/
    
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < (unsigned int) imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }
    
    /*****************************CUDA*****************************/
    
    CUDAsafeCall(hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice));
    CUDAsafeCall(hipDeviceSynchronize());
    CUDAsafeCall(hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice));
    
    /**************************************************************/
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles


    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  

    int blockDimX = imageW;
    
    dim3 dimBlock(blockDimX, blockDimX);
    dim3 dimGrid(1,1);
    
    convolutionRowGPU<<<dimGrid, dimBlock>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    CUDAsafeCall(hipPeekAtLastError());
    
    CUDAsafeCall(hipDeviceSynchronize());
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    CUDAsafeCall(hipPeekAtLastError());
    
    CUDAsafeCall(hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost));


    int err;
    float acc;
    for(acc = maxAccuracy; acc <= minAccuracy; acc *= 10) {
	err = 0;
	for(i = 0; i < (unsigned int) imageW*imageH; i++) {
	    if(acc < ABS(h_OutputCPU[i] - h_OutputGPU[i])) {
		err = 1;
		break;
	    }
	}
	
	if(err == 0) {
	    printf("Max Accuracy: %f\n", acc);
	    break;
	}
    }
    
    if(err)
	printf("Image is not accurate with filter: %i x %x\n", filter_radius, filter_radius);
    
    
    CUDAsafeCall(hipFree(d_Filter));
    CUDAsafeCall(hipFree(d_Buffer));
    CUDAsafeCall(hipFree(d_Input));
    CUDAsafeCall(hipFree(d_OutputGPU));
    
    free(h_OutputGPU);
    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    CUDAsafeCall(hipDeviceReset());


    return 0;
}
